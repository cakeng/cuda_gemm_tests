#include "hip/hip_runtime.h"
#include "mat_mul.h"

#include <hip/hip_runtime.h>
#include <cstdio>

#define _BLOCK_K_SIZE 16
#define _BLOCK_M_SIZE 128
#define _BLOCK_N_SIZE 128
#define _THREAD_M_SIZE 8
#define _THREAD_N_SIZE 8
#define _THREAD_NUM ((_BLOCK_M_SIZE / _THREAD_M_SIZE) * (_BLOCK_N_SIZE / _THREAD_N_SIZE))
#define _CACHE_A_K_PER_LOAD (_THREAD_NUM / _BLOCK_M_SIZE)
#define _CACHE_B_K_PER_LOAD (_THREAD_NUM / _BLOCK_N_SIZE)

static float *a_d, *b_d, *c_d;

__global__ void sgemm(const float *A, const float *B, float *C, const int M, const int N, const int K)
{
    const int mLocal = threadIdx.x*_THREAD_M_SIZE;
    const int nLocal = threadIdx.y*_THREAD_N_SIZE; 
    const int mGroup = blockIdx.x*_BLOCK_M_SIZE;
    const int nGroup = blockIdx.y*_BLOCK_N_SIZE;
    const int id = threadIdx.x*(_BLOCK_N_SIZE / _THREAD_N_SIZE) + threadIdx.y;
    __shared__ float ACache [_BLOCK_K_SIZE*_BLOCK_M_SIZE];
    __shared__ float BCache [_BLOCK_K_SIZE*_BLOCK_N_SIZE];
    float cout[_THREAD_N_SIZE][_THREAD_M_SIZE];
    for (int vecN = 0; vecN < _THREAD_N_SIZE; vecN++)
    {
        for (int vecM = 0; vecM < _THREAD_M_SIZE; vecM++)
        {
            cout[vecN][vecM] = 0;
        }   
    }

    // printf ("Thread %d: %3.3f\n", id, cout[0][0]);

    int kIdx = 0;  
    if (K%_BLOCK_K_SIZE)
    {
        // Load caches.
        for (int aIdx = 0; aIdx < (_BLOCK_K_SIZE/_CACHE_A_K_PER_LOAD); aIdx++)
        {
            const int cache_idx = id*(_BLOCK_K_SIZE/_CACHE_A_K_PER_LOAD) + aIdx;
            ACache[cache_idx] = A[K*(mGroup + cache_idx%_BLOCK_M_SIZE) + kIdx + cache_idx/_BLOCK_M_SIZE];
        }
        for (int bIdx = 0; bIdx < (_BLOCK_K_SIZE/_CACHE_B_K_PER_LOAD); bIdx++)
        {
            const int cache_idx = id*(_BLOCK_K_SIZE/_CACHE_B_K_PER_LOAD) + bIdx;
            BCache[cache_idx] = B[(nGroup + cache_idx%_BLOCK_N_SIZE) + N*(kIdx + cache_idx/_BLOCK_N_SIZE)];
        }
        __syncthreads();
        // printf ("Thread %d: %3.3f\n", id, cout[0][0]);
        for (; kIdx < K%_BLOCK_K_SIZE; kIdx++)
        {
            // Calculate.
            for (int vecN = 0; vecN < _THREAD_N_SIZE; vecN++)
            {
                for (int vecM = 0; vecM < _THREAD_M_SIZE; vecM++)
                {
                    // printf ("B%dT%d: (%d, %d) %3.3f, %3.3f\n", blockIdx.x + blockIdx.y
                    //     ,id, vecN, vecM, ACache[kk*_BLOCK_M_SIZE + mLocal + vecM], BCache[kk*_BLOCK_N_SIZE + nLocal + vecN]);
                    cout[vecN][vecM] += ACache[kIdx*_BLOCK_M_SIZE + mLocal + vecM] * BCache[kIdx*_BLOCK_N_SIZE + nLocal + vecN];
                }   
            }
        }
        // Sync threads.
        __syncthreads();
        // printf ("Thread %d: %3.3f\n", id, cout[0][0]);
    }
    for (; kIdx < K; kIdx += _BLOCK_K_SIZE)
    {
        // Load caches.
        for (int aIdx = 0; aIdx < (_BLOCK_K_SIZE/_CACHE_A_K_PER_LOAD); aIdx++)
        {
            const int cache_idx = id*(_BLOCK_K_SIZE/_CACHE_A_K_PER_LOAD) + aIdx;
            ACache[cache_idx] = A[K*(mGroup + cache_idx%_BLOCK_M_SIZE) + kIdx + cache_idx/_BLOCK_M_SIZE];
        }
        for (int bIdx = 0; bIdx < (_BLOCK_K_SIZE/_CACHE_B_K_PER_LOAD); bIdx++)
        {
            const int cache_idx = id*(_BLOCK_K_SIZE/_CACHE_B_K_PER_LOAD) + bIdx;
            BCache[cache_idx] = B[(nGroup + cache_idx%_BLOCK_N_SIZE) + N*(kIdx + cache_idx/_BLOCK_N_SIZE)];
        }
        __syncthreads();
        for (int kk = 0; kk < _BLOCK_K_SIZE; kk++)
        {
            // Calculate.
            for (int vecN = 0; vecN < _THREAD_N_SIZE; vecN++)
            {
                for (int vecM = 0; vecM < _THREAD_M_SIZE; vecM++)
                {
                    cout[vecN][vecM] += ACache[kk*_BLOCK_M_SIZE + mLocal + vecM] * BCache[kk*_BLOCK_N_SIZE + nLocal + vecN];
                }   
            }
        }
        // Sync threads.
        __syncthreads();
    }
    // Save results
    const int m = mGroup + mLocal;
    const int n = nGroup + nLocal;
    for (int vecN = 0; vecN < _THREAD_N_SIZE; vecN++)
    {
        for (int vecM = 0; vecM < _THREAD_M_SIZE; vecM++)
        {
            if (m + vecM < M &&  n + vecN < N)
                C[(n + vecN) + N*(m + vecM)] = cout[vecN][vecM];
        }   
    }
}

void mat_mul_write_to_gpu(float *A, float *B, float *C, int M, int N, int K)
{
    hipMemcpyAsync(a_d, A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyAsync(b_d, B, K * N * sizeof(float), hipMemcpyHostToDevice);
}

void mat_mul_read_from_gpu(float *A, float *B, float *C, int M, int N, int K)
{
    hipMemcpyAsync(C, c_d, M * N * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
}

void mat_mul(float *A, float *B, float *C, int M, int N, int K, int skip_data_movement)
{
    if (!skip_data_movement)
        mat_mul_write_to_gpu (A, B, C, M, N, K);
    
    dim3 gridDim (M/_BLOCK_M_SIZE + ((M%_BLOCK_M_SIZE) > 0), N/_BLOCK_N_SIZE + ((N%_BLOCK_N_SIZE) > 0), 1);
    dim3 blockDim ((_BLOCK_M_SIZE / _THREAD_M_SIZE), (_BLOCK_N_SIZE / _THREAD_N_SIZE), 1);
    sgemm<<<gridDim, blockDim>>>(a_d, b_d, c_d, M, N, K);
    
    if (!skip_data_movement)
        mat_mul_read_from_gpu (A, B, C, M, N, K);
}

void mat_mul_init(float *A, float *B, float *C, int M, int N, int K)
{
    printf ("Block Settings: M: %d, N: %d, K: %d, VecM: %d, VecN: %d, Thread Num: %d, ACache: %d, BCache: %d\n",
        _BLOCK_M_SIZE, _BLOCK_N_SIZE, _BLOCK_K_SIZE, _THREAD_M_SIZE, _THREAD_N_SIZE, _THREAD_NUM, _BLOCK_K_SIZE*_BLOCK_M_SIZE, _BLOCK_K_SIZE*_BLOCK_N_SIZE);
    printf ("Num blocks: %d\n", (_BLOCK_M_SIZE / _THREAD_M_SIZE)*(_BLOCK_N_SIZE / _THREAD_N_SIZE));
    if (!(((_BLOCK_K_SIZE*_BLOCK_M_SIZE)%_THREAD_NUM) == 0 && ((_BLOCK_K_SIZE*_BLOCK_N_SIZE)%_THREAD_NUM) == 0))
    {
        printf ("ERROR! - Wrong parameter settings.\n"); 
        exit(0);
    }
    
    int num_devices;
    hipGetDeviceCount(&num_devices);
    printf("Using %d devices\n", num_devices);
    for (int i = 0; i < num_devices; i++)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("[GPU %d] %s\n", i, prop.name);
    }

    if (num_devices <= 0)
    {
        printf("No CUDA device found. Aborting\n");
        exit(1);
    }

    hipMalloc(&a_d, (M+_BLOCK_M_SIZE) * (K+_BLOCK_K_SIZE) * sizeof(float));
    hipMalloc(&b_d, (K+_BLOCK_K_SIZE) * (N+_BLOCK_N_SIZE) * sizeof(float));
    hipMalloc(&c_d, M * N * sizeof(float));
    hipDeviceSynchronize();
}